#include "hip/hip_runtime.h"
/*
 * Simple example: combines data from two existing datasets using
 * CUDA and GPUDirect Storage I/O.
 *
 * To embed it in an existing HDF5 file, run:
 * $ hdf5-udf example-add_datasets.h5 example-add_datasets.cu
 *
 */
#include <math.h>

__global__ void add(int *a, int *b, int *out, size_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int n2 = (2 * a[i] + 1) * (2 * b[i] + 1);
        out[i] = (2 * a[i] + 1 - sqrt((float) (n2 - 8 * (a[i] - b[i])))) / 2;
    }
}

extern "C" void dynamic_dataset()
{
    auto ds1_data = lib.getData<int>("Dataset1");
    auto ds2_data = lib.getData<int>("Dataset2");
    auto udf_data = lib.getData<int>("UserDefinedDataset");
    auto udf_dims = lib.getDims("UserDefinedDataset");

    size_t n = udf_dims[0] * udf_dims[1];
    int block_size = 1024;
    int grid_size = (int) ceil((float) (n * sizeof(int))/block_size);
    add<<<grid_size, block_size>>>(ds1_data, ds2_data, udf_data, n);
}
